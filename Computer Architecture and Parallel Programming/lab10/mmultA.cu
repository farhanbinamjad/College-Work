/*  Programmer:  Mark Fienup
    File:        mmultA.cu
    Load Module: module load cudatoolkit
    Compile As:  nvcc -o mmultA mmultA.cu
    Run As:      qsub qsub.mmultA
    Description:  A CUDA solution to the matrix multiplication
    Stores matrices in 1-D arrays in row-order.
*/

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16
#define TRUE 1
#define FALSE 0
#define BOOL int

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
  if (err != hipSuccess) {
    printf( "%s in %s at line %d\n", hipGetErrorString( err ),
	    file, line );
    exit( EXIT_FAILURE );
  }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

typedef struct {
  int width;
  int height;
  float * elements;
} Matrix;

// function prototypes
Matrix initializeMatrix(int rows, int columns, float min, float max);
void printMatrix(Matrix M);
BOOL equalMatrices(const Matrix M1, const Matrix M2, float tolerance);
void seqMatrixMult(const Matrix A, const Matrix B, Matrix C);

__global__ void matrixMultKernel(const Matrix A, const Matrix B, Matrix C);

int main(int argc, char* argv[]) {
  Matrix A, B, C, seqC;
  int n;  // assume square
  size_t size;

  hipDeviceProp_t prop;
  // fermi1 device #0 is Tesla C2070 has 2.0 Compute Capability
  HANDLE_ERROR(hipGetDeviceProperties( &prop, 0));
  HANDLE_ERROR(hipSetDevice(0));  
  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  if (argc != 2) {
     printf("Usage: %s <matrix size>\n", argv[0]);
     exit(-1);  
  } // end if 

  sscanf(argv[1], "%d", &n);
  printf("n = %d\n",n);
  
  A = initializeMatrix(n, n, +1.0, -1.0);
  B = initializeMatrix(n, n, +1.0, -1.0);
  C = initializeMatrix(n, n, +1.0, -1.0);
  seqC = initializeMatrix(n, n, +1.0, -1.0);

  /* Do the actual work sequentially */
  hipEventRecord(start,0);

  seqMatrixMult(A, B, seqC);

  hipEventRecord(stop, 0);
  hipEventSynchronize( stop);
  float elapsedTime;
  hipEventElapsedTime( &elapsedTime, start, stop);
  printf( "Time perform seq. Matrix Multiplication on host: %3.1f ms\n", elapsedTime);
  
  // Do the work on GPU 
  hipEventRecord(start,0);
  Matrix d_A, d_B, d_C;

  // Copy A and B to device memory
  d_A = A;
  size = A.width * A.height * sizeof(float);
  hipMalloc((void**) &d_A.elements, size);
  hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);

  d_B = B;
  size = B.width * B.height * sizeof(float);
  hipMalloc((void**) &d_B.elements, size);
  hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

  // Allocate storage for C in device memory
  d_C = C;
  size = C.width * C.height * sizeof(float);
  hipMalloc((void**) &d_C.elements, size);
  
  // Set-up dimensions of blocks and grid
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid((C.width + BLOCK_SIZE - 1) / BLOCK_SIZE, (C.height + BLOCK_SIZE - 1) / BLOCK_SIZE );

  // Invoke kernel
  matrixMultKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
  
  // Copy GPU calculated C back to host memory
  hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
  
  hipEventRecord(stop, 0);
  hipEventSynchronize( stop);
  
  hipEventElapsedTime( &elapsedTime, start, stop);
  printf( "Time perform Matrix Multiplication on CUDA device: %3.1f ms\n", elapsedTime);
  
  if (equalMatrices(C, seqC, 0.001)) {
    printf("Results match within a tolerance of %f\n", 0.001);
  } else {
    printf("Results wrong:  tolerance used %f\n", 0.001);
  } // end if

  // print if small enough
  if (C.width < 10 && C.height < 10) {
     printf("\nC from CUDA:\n");
     printMatrix(C);
     printf("\nseqC from host:\n");
     printMatrix(seqC);
  } // end if  

  hipEventDestroy( start );
  hipEventDestroy( stop );
  hipFree( d_A.elements );
  hipFree( d_B.elements );
  hipFree( d_C.elements );
  
  free(A.elements);
  free(B.elements);
  free(C.elements);
  free(seqC.elements);
  
  return 0;
} /* end main */

// Each thread computes one element of C by accumulating results in local CValue
__global__ void matrixMultKernel(const Matrix A, const Matrix B, Matrix C) {
  int row = threadIdx.y + blockIdx.y * blockDim.y;
  int col = threadIdx.x + blockIdx.x * blockDim.x;
  int k;
  float CValue;

  if (row < C.height && col < C.width) {
     CValue = 0.0;
     for (k = 0; k < A.width; k++) {
     	 CValue += A.elements[row * A.width + k] * B.elements[k * B.width + col];
     } // end for
     C.elements[row * C.width + col] = CValue;
  } // end if
} // end matrixMultKernel


/***************************************************************
 Allocate matrix of floats at 1D array and initialized elements
 randomly.
 ***************************************************************/
Matrix initializeMatrix(int rows, int columns, float min, float max) {
  int i;
  float range, div;
  Matrix M;

  M.width = columns;
  M.height = rows;
  M.elements = (float *) malloc(sizeof(float) * rows * columns);

  for (i=0; i < rows*columns; i++) {
      range = max - min;
      div = RAND_MAX / range;
      M.elements[i] = min + (rand() / div);
  } // end for

  return M;
} // end initializeMatrix


/***************************************************************
 Prints matrix to screen
****************************************************************/
void printMatrix(Matrix M) {
  int r, c;

  for (r=0; r < M.height; r++) {
    for (c=0; c < M.width; c++) {
    	printf("%8.4f ",M.elements[r * M.width + c]);
    } // end for (c... 
    printf("\n");
  } // end for (r... 

} // end printMatrix


/***************************************************************
 Compares elements of M1 and M2 to see whether all match within
 the given tolerance.  Returns TRUE or FALSE accordingly.
****************************************************************/
BOOL equalMatrices(const Matrix M1, const Matrix M2, float tolerance) {
  int i;

  if (M1.width != M2.width || M1.height != M2.height) {
     return FALSE;
  } // end if

  for (i=0; i < M1.width*M1.height; i++) {
      if (fabsf(M1.elements[i] - M2.elements[i]) > tolerance) {
printf("MISMATCH VALUES: %f %f\n", M1.elements[i],M2.elements[i]);
        return FALSE;
      } // end if
   } // end for
   return TRUE;

} // end equalMatrices


/***************************************************************
 Sequentially computes matrix multiplication of C = A * B with
 C being returned.
****************************************************************/
void seqMatrixMult(const Matrix A, const Matrix B, Matrix C) {
  int i, j, k;
  float CValue;

  for (i=0; i < C.height; i++) {
    for (j=0; j < C.width; j++) {
      CValue = 0.0;
      for (k=0; k < A.width; k++) {
        CValue += A.elements[i*A.width + k] * B.elements[k*B.width + j];
      } /* end for (k */
      C.elements[i*C.width + j] = CValue;
    } /* end for (j */
  } /* end for (i */
  
} // end seqMatrixMult
