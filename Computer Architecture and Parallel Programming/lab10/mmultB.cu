/*  Programmer:  Mark Fienup
    File:        mmultB.cu
    Compile As:  nvcc -o mmultB mmultB.cu
    Run As:      ./mmultB <matrix size>
    Description:  A CUDA solution to the matrix multiplication
    Stores matrices in 1-D arrays in row-order.
*/

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16
#define TRUE 1
#define FALSE 0
#define BOOL int

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
  if (err != hipSuccess) {
    printf( "%s in %s at line %d\n", hipGetErrorString( err ),
	    file, line );
    exit( EXIT_FAILURE );
  }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

typedef struct {
  int width;
  int height;
  int stride;
  float * elements;
} Matrix;


// Get a matrix element
__device__ float GetElement(const Matrix A, int row, int col) {
  return A.elements[row * A.stride + col];
} // end GetElement

// Set a matrix element
__device__ void SetElement(Matrix A, int row, int col, float value) {
  A.elements[row * A.stride + col] = value;
} // end SetElement

// Get the BLOCK_SIZE x BLOCK_SIZE sub-matrix Asub of A that is 
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of A
__device__ Matrix GetSubMatrix(Matrix A, int row, int col) {

  Matrix Asub;

  Asub.width = BLOCK_SIZE;
  Asub.height = BLOCK_SIZE;
  Asub.stride = A.stride;
  Asub.elements = &A.elements[A.stride*BLOCK_SIZE*row + BLOCK_SIZE*col];

  return Asub;
} // end GetSubMatrix

// function prototypes
Matrix initializeMatrix(int rows, int columns, float min, float max);
void printMatrix(Matrix M);
BOOL equalMatrices(const Matrix M1, const Matrix M2, float tolerance);
void seqMatrixMult(const Matrix A, const Matrix B, Matrix C);

__global__ void matrixMultKernel(const Matrix A, const Matrix B, Matrix C);


int main(int argc, char* argv[]) {
  Matrix A, B, C, seqC;
  int n;  // assume square
  size_t size;

  hipDeviceProp_t prop;
  // fermi1 device #0 is Tesla C2070 has 2.0 Compute Capability
  HANDLE_ERROR(hipGetDeviceProperties( &prop, 0));
  HANDLE_ERROR(hipSetDevice(0));  
  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  if (argc != 2) {
     printf("Usage: %s <matrix size>\n", argv[0]);
     exit(-1);  
  } // end if 

  sscanf(argv[1], "%d", &n);
  printf("n = %d\n",n);
  
  A = initializeMatrix(n, n, +1.0, -1.0);
  B = initializeMatrix(n, n, +1.0, -1.0);
  C = initializeMatrix(n, n, +1.0, -1.0);
  seqC = initializeMatrix(n, n, +1.0, -1.0);

  /* Do the actual work sequentially */
  hipEventRecord(start,0);

  seqMatrixMult(A, B, seqC);

  hipEventRecord(stop, 0);
  hipEventSynchronize( stop);
  float elapsedTime;
  hipEventElapsedTime( &elapsedTime, start, stop);
  printf( "Time perform seq. Matrix Multiplication on host: %3.1f ms\n", elapsedTime);
  
  // Do the work on GPU 
  hipEventRecord(start,0);
  Matrix d_A, d_B, d_C;

  // Copy A and B to device memory
  d_A = A;
  d_A.stride = A.width;
  size = A.width * A.height * sizeof(float);
  hipMalloc((void**) &d_A.elements, size);
  hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);

  d_B = B;
  d_B.stride = B.width;
  size = B.width * B.height * sizeof(float);
  hipMalloc((void**) &d_B.elements, size);
  hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

  // Allocate storage for C in device memory
  d_C = C;
  d_C.stride = C.width;
  size = C.width * C.height * sizeof(float);
  hipMalloc((void**) &d_C.elements, size);
  
  // Set-up dimensions of blocks and grid
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid((B.width + BLOCK_SIZE - 1) / BLOCK_SIZE, (A.height + BLOCK_SIZE - 1) / BLOCK_SIZE );

  // Invoke kernel
  matrixMultKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
  
  // Copy GPU calculated C back to host memory
  hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
  
  hipEventRecord(stop, 0);
  hipEventSynchronize( stop);
  
  hipEventElapsedTime( &elapsedTime, start, stop);
  printf( "Time perform Matrix Multiplication on CUDA device: %3.1f ms\n", elapsedTime);
  
  if (equalMatrices(C, seqC, 0.001)) {
    printf("Results match within a tolerance of %f\n", 0.001);
  } else {
    printf("Results wrong:  tolerance used %f\n", 0.001);
  } // end if

  // print if small enough
  if (C.width < 10 && C.height < 10) {
     printf("\nC from CUDA:\n");
     printMatrix(C);
     printf("\nseqC from host:\n");
     printMatrix(seqC);
  } // end if  

  hipEventDestroy( start );
  hipEventDestroy( stop );
  hipFree( d_A.elements );
  hipFree( d_B.elements );
  hipFree( d_C.elements );
  
  free(A.elements);
  free(B.elements);
  free(C.elements);
  free(seqC.elements);
  
  return 0;
} /* end main */

// Each thread computes one element of C by accumulating results in local CValue
__global__ void matrixMultKernel(const Matrix A, const Matrix B, Matrix C) {

  // block row and column
  int blockRow = blockIdx.y;
  int blockCol = blockIdx.x;

  // Each thread block computes one sub-matrix Csub of C
  Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

  // Each thread computes one element of Csub by
  // accumulating results into CValue
  float CValue = 0.0;

  // Thread row and column within Csub
  int row = threadIdx.y;
  int col = threadIdx.x;

  // Loop over all the sub-matrices of A and B that are required
  // to computer Csub
  // Multiply each pair of sub-matrices together and accumulate
  // the results
  for (int m=0; m < (A.width/BLOCK_SIZE); m++) {

     // Get sub-matrix of A
     Matrix Asub = GetSubMatrix(A, blockRow, m);

     // Get sub-matrix of B
     Matrix Bsub = GetSubMatrix(B, m, blockCol);

     // Shared memory used to store Asub and Bsub
     __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
     __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

     // Load Asub and Bsub from device memory to shared memory
     // Each thread loads one element of each sub-matrix
     As[row][col] = GetElement(Asub, row, col);
     Bs[row][col] = GetElement(Bsub, row, col);

     // Synchronize to make sure the sub-matrices are loaded 
     // before starting to computation of Csub
     __syncthreads();

     // Multiply Asub and Bsub together
     for (int k = 0; k < BLOCK_SIZE; k++) {
     	 CValue += As[row][k] * Bs[k][col];
     } // end for

     // Synchronize to make sure that the preceeding 
     // computation is done before loading two new 
     // sub-matrices of A and B in the next iteration
     __syncthreads();

  } // end for (m

  // Write Csub to device memory
  // Each thread writes one element
  SetElement(Csub, row, col, CValue);

} // end matrixMultKernel


/***************************************************************
 Allocate matrix of floats at 1D array and initialized elements
 randomly.
 ***************************************************************/
Matrix initializeMatrix(int rows, int columns, float min, float max) {
  int i;
  float range, div;
  Matrix M;

  M.width = columns;
  M.height = rows;
  M.elements = (float *) malloc(sizeof(float) * rows * columns);

  for (i=0; i < rows*columns; i++) {
      range = max - min;
      div = RAND_MAX / range;
      M.elements[i] = min + (rand() / div);
  } // end for

  return M;
} // end initializeMatrix


/***************************************************************
 Prints matrix to screen
****************************************************************/
void printMatrix(Matrix M) {
  int r, c;

  for (r=0; r < M.height; r++) {
    for (c=0; c < M.width; c++) {
    	printf("%8.4f ",M.elements[r * M.width + c]);
    } // end for (c... 
    printf("\n");
  } // end for (r... 

} // end printMatrix


/***************************************************************
 Compares elements of M1 and M2 to see whether all match within
 the given tolerance.  Returns TRUE or FALSE accordingly.
****************************************************************/
BOOL equalMatrices(const Matrix M1, const Matrix M2, float tolerance) {
  int i;

  if (M1.width != M2.width || M1.height != M2.height) {
     return FALSE;
  } // end if

  for (i=0; i < M1.width*M1.height; i++) {
      if (fabsf(M1.elements[i] - M2.elements[i]) > tolerance) {
printf("MISMATCH VALUES: %f %f\n", M1.elements[i],M2.elements[i]);
        return FALSE;
      } // end if
   } // end for
   return TRUE;

} // end equalMatrices


/***************************************************************
 Sequentially computes matrix multiplication of C = A * B with
 C being returned.
****************************************************************/
void seqMatrixMult(const Matrix A, const Matrix B, Matrix C) {
  int i, j, k;
  float CValue;

  for (i=0; i < C.height; i++) {
    for (j=0; j < C.width; j++) {
      CValue = 0.0;
      for (k=0; k < A.width; k++) {
        CValue += A.elements[i*A.width + k] * B.elements[k*B.width + j];
      } /* end for (k */
      C.elements[i*C.width + j] = CValue;
    } /* end for (j */
  } /* end for (i */
  
} // end seqMatrixMult