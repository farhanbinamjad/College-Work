/*  Programmer:  Mark Fienup
    File:        addVectors.cu
    Compile As:  nvcc -o addVectors addVectors.cu
    Run As:      qsub qsub.addVectors
    Description:  A CUDA solution to the add two 1-D vectors.
*/

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16
#define TRUE 1
#define FALSE 0
#define BOOL int

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
  if (err != hipSuccess) {
    printf( "%s in %s at line %d\n", hipGetErrorString( err ),
	    file, line );
    exit( EXIT_FAILURE );
  }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

// function prototypes
float*  initializeVector(int length, float min, float max);
void printVector(int length, float *);
BOOL equalVectors(int length, float * vector1, float * vector2, float tolerance);
void seqVectorAddition(int length, float * a, float * b, float * c);

__global__ void vectorAdditionKernel(int length, float * a, float * b, float * c);

int main(int argc, char* argv[]) {
  float * a;
  float * b;
  float * c;
  float * seqC;
  int length;  // assume square
  size_t size;

  hipDeviceProp_t prop;
  // fermi1 device #0 is Tesla C2070 has 2.0 Compute Capability
  HANDLE_ERROR(hipGetDeviceProperties( &prop, 0));
  HANDLE_ERROR(hipSetDevice(0));  
  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  if (argc != 2) {
     printf("Usage: %s <length>\n", argv[0]);
     exit(-1);  
  } // end if 

  sscanf(argv[1], "%d", &length);
  printf("vector length = %d\n",length);
  
  a = initializeVector(length, +1.0, -1.0);
  b = initializeVector(length, +1.0, -1.0);
  c = initializeVector(length, +1.0, -1.0);
  seqC = initializeVector(length, +1.0, -1.0);

  /* Do the actual work sequentially */
  hipEventRecord(start,0);

  seqVectorAddition(length, a, b, seqC);

  hipEventRecord(stop, 0);
  hipEventSynchronize( stop);
  float elapsedTime;
  hipEventElapsedTime( &elapsedTime, start, stop);
  printf( "Time perform seq. vector addition on host: %3.1f ms\n", elapsedTime);

  // Do the work on GPU 
  hipEventRecord(start,0);
  float * d_a;
  float * d_b;
  float * d_c;

  // Copy a and b to device memory
  size = length * sizeof(float);
  hipMalloc((void**) &d_a, size);
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

  hipMalloc((void**) &d_b, size);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  // Allocate storage for c in device memory
  hipMalloc((void**) &d_c, size);
  
  // Invoke kernel with 128 blocks, each with 128 threads
  vectorAdditionKernel<<<128, 128>>>(length, d_a, d_b, d_c);
  
  // Copy GPU calculated c back to host memory
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
  
  hipEventRecord(stop, 0);
  hipEventSynchronize( stop);
  
  hipEventElapsedTime( &elapsedTime, start, stop);
  printf( "Time perform vector addition on CUDA device: %3.1f ms\n", elapsedTime);
  
  if (equalVectors(length, c, seqC, 0.001)) {
    printf("Results match within a tolerance of %f\n", 0.001);
  } else {
    printf("Results wrong:  tolerance used %f\n", 0.001);
  } // end if

  // print if small enough
  if (length < 10) {
     printf("\nc from CUDA:\n");
     printVector(length, c);
     printf("\nseqC from host:\n");
     printVector(length, seqC);
  } // end if  

  hipEventDestroy( start );
  hipEventDestroy( stop );
  hipFree( d_a );
  hipFree( d_b );
  hipFree( d_c );
  
  free(a);
  free(b);
  free(c);
  free(seqC);
  
  return 0;
} /* end main */


// Each thread might compute many elements with each stride apart
__global__ void vectorAdditionKernel(int length, float * a, float * b, float * c) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = gridDim.x * blockDim.x;

  while ( tid < length ) {
     c[tid] = a[tid] + b[tid];
     tid += stride;
  } // end while
} // end vectorAdditionKernel


/***************************************************************
 Allocate vector of floats and initialized elements
 randomly.
 ***************************************************************/
float* initializeVector(int length, float min, float max) {
  int i;
  float range, div;
  float * temp;

  temp = (float *) malloc(sizeof(float) * length);

  for (i=0; i < length; i++) {
      range = max - min;
      div = RAND_MAX / range;
      temp[i] = min + (rand() / div);
  } // end for

  return temp;
} // end initializeVector


/***************************************************************
 Prints vector to screen
****************************************************************/
void printVector(int length, float * v) {
  int i;

  for (i=0; i < length; i++) {
      printf("%8.4f ", v[i]);
  } // end for
    printf("\n");
} // end printVector



/***************************************************************
 Compares elements of vector1 and vector2 to see whether all match within
 the given tolerance.  Returns TRUE or FALSE accordingly.
****************************************************************/
BOOL equalVectors(int length, float * vector1, float * vector2, float tolerance) {
  int i;

  for (i=0; i < length; i++) {
      if (fabsf(vector1[i] - vector2[i]) > tolerance) {
      	 printf("MISMATCH VALUES: %f %f\n", vector1[i], vector2[i]);
         return FALSE;
      } // end if
   } // end for
   return TRUE;

} // end equalVectors


/***************************************************************
 Sequentially computes vector addition of c = a + b with
 c being returned.
****************************************************************/
void seqVectorAddition(int length, float * a, float * b, float * c) {
  int i;

  for (i=0; i < length; i++) {
     c[i] = a[i] + b[i];
  } /* end for (i */
  
} // end seqVectorAddition
